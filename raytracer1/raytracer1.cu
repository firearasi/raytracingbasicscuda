#include "hip/hip_runtime.h"
#include <iostream>
#include "ray.h"
#define TX 32
#define TY 32
using namespace std;

__device__ vec3 color (const ray& r)
{
	vec3 unit_direction = unit_vector(r.direction());
	float t=0.5*(unit_direction.y() + 1.0);
	float s=0.5*(unit_direction.x()+1.0);
	return lerp(vec3(1.0,0.3,0.9), lerp(vec3(1.0,1.0,1.0), vec3(0.5,0.7,1.0), t), s);
}

__global__ void renderKernel(vec3* d_pixels, int nx, int ny)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x;
	int j = threadIdx.y+blockIdx.y*blockDim.y;
	if(i>=nx||j>=ny)
		return;
	vec3 lower_left_corner(-2.0,-1.0,-1.0);
	vec3 horizontal(4.0,0.0,0.0);
	vec3 vertical(0.0,2.0,0.0);
	vec3 origin(0.0,0.0,0.0);
	float u=float(i)/float(nx);
	float v=float(j)/float(ny);
	vec3 pt(lower_left_corner + u*horizontal+v*vertical);
	ray r(origin, pt);
	d_pixels[i+j*nx]=color(r);

}

int main()
{
	int nx=200;
	int ny=100;
	cout << "P3\n" << nx << " " << ny << "\n255\n";

	vec3* d_pixels;
	hipMallocManaged(&d_pixels,nx*ny*sizeof(vec3));
	dim3 blockSize(TX,TY);
	dim3 gridSize((nx+TX-1)/TX,(ny+TY-1)/TY);
	renderKernel<<<gridSize,blockSize>>>(d_pixels,nx,ny);
	hipDeviceSynchronize();
	for (int j=ny-1;j>=0;j--)
		for (int i=0;i<nx;i++)
		{
			int ir=int(255.99*d_pixels[i+j*nx].r());
			int ig=int(255.99*d_pixels[i+j*nx].g());
			int ib=int(255.99*d_pixels[i+j*nx].b());
			cout << ir<<" " << ig<<" " << ib<<"\n";
		}

	hipFree(d_pixels);
	return 0;
}
