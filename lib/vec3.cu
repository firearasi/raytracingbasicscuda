#include "hip/hip_runtime.h"
//==================================================================================================
// Written in 2016 by Peter Shirley <ptrshrl@gmail.com>
//
// To the extent possible under law, the author(s) have dedicated all copyright and related and
// neighboring rights to this software to the public domain worldwide. This software is distributed
// without any warranty.
//
// You should have received a copy (see file COPYING.txt) of the CC0 Public Domain Dedication along
// with this software. If not, see <http://creativecommons.org/publicdomain/zero/1.0/>.
//==================================================================================================

#include "vec3.h"
extern "C"{

__device__ __host__  void vec3::make_unit_vector() {
    float k = 1.0 / sqrt(e[0]*e[0] + e[1]*e[1] + e[2]*e[2]);
    e[0] *= k; e[1] *= k; e[2] *= k;
}

__device__ __host__ vec3 vec3::clamp()
{
	float rr,gg,bb;
	rr=r();
	gg=g();
	bb=b();
	if(rr<0) rr=0;
	else if(rr>1) rr=1;
	if(gg<0) gg=0;
	else if(gg>1) gg=1;
	if(bb<0) bb=0;
	else if(bb>1) bb=1;
	return vec3(rr,gg,bb);
}

__device__ __host__  vec3 operator+(const vec3 &v1, const vec3 &v2) {
    return vec3(v1.e[0] + v2.e[0], v1.e[1] + v2.e[1], v1.e[2] + v2.e[2]);
}

__device__ __host__  vec3 operator-(const vec3 &v1, const vec3 &v2) {
    return vec3(v1.e[0] - v2.e[0], v1.e[1] - v2.e[1], v1.e[2] - v2.e[2]);
}

__device__ __host__  vec3 operator*(const vec3 &v1, const vec3 &v2) {
    return vec3(v1.e[0] * v2.e[0], v1.e[1] * v2.e[1], v1.e[2] * v2.e[2]);
}

__device__ __host__  vec3 operator/(const vec3 &v1, const vec3 &v2) {
    return vec3(v1.e[0] / v2.e[0], v1.e[1] / v2.e[1], v1.e[2] / v2.e[2]);
}

__device__ __host__  vec3 mult(float t, const vec3 &v) {
    return vec3(t*v.e[0], t*v.e[1], t*v.e[2]);
}




__device__ __host__  float dot(const vec3 &v1, const vec3 &v2) {
    return v1.e[0] *v2.e[0] + v1.e[1] *v2.e[1]  + v1.e[2] *v2.e[2];
}

__device__ __host__  vec3 cross(const vec3 &v1, const vec3 &v2) {
    return vec3( (v1.e[1]*v2.e[2] - v1.e[2]*v2.e[1]),
                (-(v1.e[0]*v2.e[2] - v1.e[2]*v2.e[0])),
                (v1.e[0]*v2.e[1] - v1.e[1]*v2.e[0]));
}


__device__ __host__  vec3& vec3::operator+=(const vec3 &v){
    e[0]  += v.e[0];
    e[1]  += v.e[1];
    e[2]  += v.e[2];
    return *this;
}

__device__ __host__  vec3& vec3::operator*=(const vec3 &v){
    e[0]  *= v.e[0];
    e[1]  *= v.e[1];
    e[2]  *= v.e[2];
    return *this;
}

__device__ __host__  vec3& vec3::operator/=(const vec3 &v){
    e[0]  /= v.e[0];
    e[1]  /= v.e[1];
    e[2]  /= v.e[2];
    return *this;
}

__device__ __host__  vec3& vec3::operator-=(const vec3& v) {
    e[0]  -= v.e[0];
    e[1]  -= v.e[1];
    e[2]  -= v.e[2];
    return *this;
}

__device__ __host__  vec3& vec3::operator*=(const float t) {
    e[0]  *= t;
    e[1]  *= t;
    e[2]  *= t;
    return *this;
}

__device__ __host__  vec3& vec3::operator/=(const float t) {
    float k = 1.0/t;
    
    e[0]  *= k;
    e[1]  *= k;
    e[2]  *= k;
    return *this;
}

__device__ __host__ vec3& vec3::operator=(const vec3& v2)
{
	e[0]=v2.e[0];
	e[1]=v2.e[1];
	e[2]=v2.e[2];
	return *this;
}

__device__ __host__  vec3 unit_vector(vec3 v) {
    return mult(1.0 / v.length(),v);
}

__device__ __host__  vec3 lerp(const vec3& u, const vec3& v, float t)
{
	return mult((1.0 - t) , u) + mult(t , v);
}

__device__ __host__  vec3 reflect(const vec3& v, const vec3&n)
{
	return v- mult(2*dot(v,n),n);
}

__device__ __host__  bool refract(const vec3&v, const vec3&n, float ni_over_nt, vec3& refracted)
{
	vec3 uv = unit_vector(v);
	float dt = dot(uv, n);
	float discriminant = 1.0 - ni_over_nt * ni_over_nt*(1-dt*dt);
	if(discriminant > 0)
	{
		refracted = mult(ni_over_nt , (uv-mult(dt,n))) -  mult(sqrt(discriminant),n);
		return true;
	}
	else
		return false;
}

__device__ __host__  vec3 random_in_unit_sphere()
{
    vec3 p;
    do {
    	p = mult(2.0 , vec3(drand48(),drand48(),drand48())-vec3(1,1,1));
    } while(p.squared_length()>=1.0);
    return p;
}

__device__ __host__  vec3 random_in_unit_disk()
{
    vec3 p;
    do {
    	p = mult(2.0 , vec3(drand48(),drand48(),0)-vec3(1,1,0));
    } while(p.squared_length()>=1.0);
	return p;
}
                                                
}
